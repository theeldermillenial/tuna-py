#include "hip/hip_runtime.h"
// #define VERIFY_HASH		//Execute only 1 thread and verify manually

#include <cstdio>
#include <cstdlib>
#include <stdbool.h>
#include <stdint.h>
#include <random>

#include "cuPrintf.cu"
#include "cuPrintf.cuh"
extern "C" {
	#include "sha256.h"
	#include "utils.h"
}
#include "sha256_unrolls.h"

#include <chrono>
#include <iomanip>
#include <iostream>
#include <thread>
#include <string>
#include <vector>

#ifndef VERIFY_HASH
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>
#include <pybind11/complex.h>

namespace py = pybind11;
#endif


//#define ITERATE_BLOCKS	//Don't define BDIMX and create a 65535x1 Grid

/*
	Threads = BDIMX*GDIMX*GDIMY
	Thread Max = 2^32
	The most convenient way to form dimensions is to use a square grid of blocks
	GDIMX = sqrt(2^32/BDIMX)
*/
#ifndef VERIFY_HASH
#define BDIMX		256		//MAX = 512
#define GDIMX		32		//MAX = 65535 = 2^16-1
#define GDIMY		GDIMX
#endif

#ifdef VERIFY_HASH
#define BDIMX	1
#define GDIMX	1
#define GDIMY	1
#endif

__global__ void kernel_sha256d(unsigned int *nr, void *debug);


__constant__ unsigned char device_data[105];
__constant__ unsigned char device_difficulty[16];
__constant__ unsigned long device_msg_len;
__constant__ unsigned long nloops;

inline void gpuAssert(hipError_t code, char *file, int line, bool abort)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"CUDA_SAFE_CALL: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__, true); }

void hash_to_string(unsigned char * buff, unsigned long len) {
	int k, i;
	for (i = 0, k = 0; i < len; i++, k+= 2)
	{
		printf("%02x", buff[i]);
	}
}

//Warning: This mmodifies the nonce value of data so do it last!
void compute_and_print_hash(const unsigned char *data, unsigned int *nonce, unsigned long MSG_SIZE) {
	unsigned char hash[32];
	SHA256_CTX ctx;
	int i;

	printf("MSG_SIZE: %lu\n", MSG_SIZE);
	printf("Original Data: ");

	*((unsigned int *) (data + 4)) = nonce[0];
	*((unsigned int *) (data + 8)) = nonce[1];
	*((unsigned int *) (data + 12)) = nonce[2];
	*((unsigned int *) (data + 16)) = nonce[3];
	hash_to_string((unsigned char *) data, MSG_SIZE);
	printf("\n");

	printf("Nonce: ");
	printf("%.8x ", nonce[0]);
	printf("%.8x ", nonce[1]);
	printf("%.8x ", nonce[2]);
	printf("%.8x ", nonce[3]);
	printf("\n");

	sha256_init(&ctx);
	sha256_update(&ctx, data, MSG_SIZE);
	sha256_final(&ctx, hash);
	sha256_init(&ctx);
	sha256_update(&ctx, hash, 32);
	sha256_final(&ctx, hash);

	printf("Hash is:\n");
	for(i=0; i<8; i++) {
		printf("%.8x ", ENDIAN_SWAP_32(*(((unsigned int *) hash) + i)));
	}
	printf("\n");
}

bool check_file(char * fname) {
	FILE * f = 0;

	f = fopen(fname, "rb");
	if (!f){
		return false;
	} else {
		return true;
	}
}

// Function to convert a hex character to its corresponding integer value
int hexCharToInt(char c) {
    if (c >= '0' && c <= '9') {
        return c - '0';
    } else if (c >= 'a' && c <= 'f') {
        return c - 'a' + 10;
    } else if (c >= 'A' && c <= 'F') {
        return c - 'A' + 10;
    }
    return -1; // Invalid hex character
}

// Function to convert a hex string to a byte array
unsigned char* hexStringToByteArray(unsigned char* hexString, unsigned long strLength) {

    // Check if the input string length is odd (invalid hex string)
    if (strLength % 2 != 0) {
        return NULL;
    }

    size_t arrayLength = strLength / 2;
    unsigned char* byteArray = (unsigned char*)malloc(arrayLength);

    for (size_t i = 0; i < arrayLength; ++i) {
        int highNibble = hexCharToInt(hexString[i * 2]);
        int lowNibble = hexCharToInt(hexString[i * 2 + 1]);

        // Check for invalid characters in the hex string
        if (highNibble == -1 || lowNibble == -1) {
            free(byteArray);
            return NULL;
        }

        byteArray[i] = (unsigned char)((highNibble << 4) | lowNibble);
    }

    return byteArray;
}

void store_nonce(char * fname, unsigned int * nonce) {

	FILE * f = 0;

	f = fopen(fname, "w");
	
	fprintf(f, "%.8x", ENDIAN_SWAP_32(nonce[0]));
	fprintf(f, "%.8x", ENDIAN_SWAP_32(nonce[1]));
	fprintf(f, "%.8x", ENDIAN_SWAP_32(nonce[2]));
	fprintf(f, "%.8x", ENDIAN_SWAP_32(nonce[3]));
	fclose(f);
}

unsigned char * get_file_data(char * fname, unsigned long * MSG_SIZE) {

	FILE * f = 0;
	unsigned char * buffer = 0;
	unsigned long fsize = 0;

	f = fopen(fname, "rb");
	while (!check_file(fname)){
		printf("Waiting for new datum...\n");
		std::this_thread::sleep_for(std::chrono::milliseconds(100));
		f = fopen(fname, "rb");
	}
	fflush(f);

	if (fseek(f, 0, SEEK_END)){
		fprintf(stderr, "Unable to fseek %s\n", fname);
		return 0;
	}
	fflush(f);
	fsize = ftell(f);
	rewind(f);
	*MSG_SIZE = fsize / 2;
	fprintf(stdout, "MSG_SIZE: %i\n", *MSG_SIZE);

	buffer = (unsigned char *)malloc((fsize+1)*sizeof(unsigned char));
	// checkCudaErrors(hipMallocManaged(&buffer, (fsize+1)*sizeof(char)));
	fread(buffer, fsize, 1, f);
	fclose(f);

	return hexStringToByteArray(buffer, fsize);
	// return buffer;
}



unsigned char * set_tuna_difficulty(unsigned short difficulty_number, unsigned char leading_zeros) {
	int i;
	unsigned char * difficulty = (unsigned char *) malloc(sizeof(unsigned char) * 16);
	for(i=0; i<16; i++) {
		difficulty[i] = 0;
	}

	int byte_location = leading_zeros / 2;
    if (leading_zeros % 2 == 0) {
        difficulty[byte_location] = (difficulty_number / 256);
        difficulty[byte_location + 1] = (difficulty_number % 256);
    } else {
        difficulty[byte_location] = (difficulty_number / 4096);
        difficulty[byte_location + 1] = ((difficulty_number / 16) % 4096);
        difficulty[byte_location + 2] = (difficulty_number % 16);
    }

	return difficulty;
}

int main(int argc, char **argv) {
	int i, j;
	// unsigned char *data = test_block;
	unsigned long MSG_SIZE;
	#ifndef VERIFY_HASH
	const unsigned char *data = get_file_data("./datum.txt", &MSG_SIZE);
	#else
	unsigned char *data = get_file_data("./datum.txt", &MSG_SIZE);
	#endif


	/*
		Host Side Preprocessing
		The goal here is to prepare and compute everything that will be shared by all threads.
	*/
	
	//Initialize Cuda stuff
	cudaPrintfInit();
	dim3 DimGrid(GDIMX,GDIMY);
	#ifndef ITERATE_BLOCKS
	dim3 DimBlock(BDIMX,1);
	#endif

	//Used to store a nonce if a block is mined
	unsigned int * host_nonce = new unsigned int[40];
	memset(host_nonce, 0, sizeof(unsigned int) * 40);

	std::mt19937 mt{ std::random_device{}() };
		

	while (true) {
		
		//Increment the global nonce
		host_nonce[0] = *((unsigned int *) (data + 4));
		host_nonce[1] = mt();
		host_nonce[2] = mt();
		host_nonce[3] = 0;

		//Decodes and stores the difficulty in a 16-byte array for convenience
		unsigned char * difficulty = set_tuna_difficulty(65535, 8);

		//Data buffer for sending debug information to/from the GPU
		unsigned char debug[32];
		unsigned char *d_debug;
		#ifdef VERIFY_HASH
		printf("Initial Data: ");
		for(i=0; i<MSG_SIZE; i++) {
			printf("%.2x", data[i]);
		}
		printf("\n");
		SHA256_CTX verify;
		sha256_init(&verify);
		printf("1. init state: ");
		for(i=0; i<8; i++) {
			printf("%.8x ", ENDIAN_SWAP_32(verify.state[i]));
		}
		printf("\n");
		sha256_update(&verify, (unsigned char *) data, MSG_SIZE);
		printf("2. update state: ");
		for(i=0; i<8; i++) {
			printf("%.8x ", ENDIAN_SWAP_32(verify.state[i]));
		}
		printf("\n");
		sha256_final(&verify, debug);
		printf("3. final state: ");
		for(i=0; i<8; i++) {
			printf("%.8x ", ENDIAN_SWAP_32(verify.state[i]));
		}
		printf("\n");
		sha256_init(&verify);
		printf("4. init state: ");
		for(i=0; i<8; i++) {
			printf("%.8x ", ENDIAN_SWAP_32(verify.state[i]));
		}
		printf("\n");
		sha256_update(&verify, (unsigned char *) debug, 32);
		printf("5. update state: ");
		for(i=0; i<8; i++) {
			printf("%.8x ", ENDIAN_SWAP_32(verify.state[i]));
		}
		printf("\n");
		sha256_final(&verify, debug);
		printf("6. final state: ");
		for(i=0; i<8; i++) {
			printf("%.8x ", ENDIAN_SWAP_32(verify.state[i]));
		}
		printf("\n");
		printf("Final Hash: ");
		for(i=0; i<8; i++) {
			printf("%.8x ", ENDIAN_SWAP_32(*(((unsigned int *) debug) + i)));
		}
		printf("\n");
		#endif

		// Copy debug data to device
		hipGetErrorString(hipMalloc((void **)&d_debug, 32*sizeof(unsigned char)));
		hipGetErrorString(hipMemcpy(d_debug, (void *) &debug, 32*sizeof(unsigned char), hipMemcpyHostToDevice));

		//Allocate space on Global Memory
		// SHA256_CTX *d_ctx;
		unsigned int * device_nonce = new unsigned int[40];
		memset(device_nonce, 0, sizeof(unsigned int) * 40);
		CUDA_SAFE_CALL(hipMalloc((void **) &device_nonce, 40 * sizeof(unsigned int)));

		/*
			Kernel Execution
			Measure and launch the kernel and start mining
		*/
		//Copy constants to device
		#ifdef VERIFY_HASH
		unsigned long NLOOPS = 1;
		#else
		unsigned long NLOOPS = 4096;
		#endif
		CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(device_data), &data[0], 105));
		CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(device_difficulty), &difficulty[0], 16));
		CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(device_msg_len), &MSG_SIZE, 4));
		CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(nloops), &NLOOPS, 4));

		// Copy nonce to device
		CUDA_SAFE_CALL(hipMemcpy(device_nonce, &host_nonce[0], 40 * sizeof(unsigned int), hipMemcpyHostToDevice));

		
		float elapsed_gpu;
		long long int num_hashes;
		#ifdef ITERATE_BLOCKS
		//Try different block sizes
		for(i=1; i <= 512; i++) {
			dim3 DimBlock(i,1);
		#endif
			//Start timers
			hipEvent_t start, stop;
			hipEventCreate(&start);
			hipEventCreate(&stop);
			hipEventRecord(start, 0);

			//Launch Kernel
			kernel_sha256d<<<DimGrid, DimBlock>>>(device_nonce, (void *) d_debug);
			
			#ifndef VERIFY_HASH
			if (check_file("./datum.txt")) {
			#else
			if (check_file("./datum.txt")) {
			#endif
				printf("Found new datum!\n");
				data = get_file_data("./datum.txt", &MSG_SIZE);
				#ifndef VERIFY_HASH
				unsigned char *data = get_file_data("./datum.txt", &MSG_SIZE);
				remove("./datum.txt");
				#else
				unsigned char *data = get_file_data("./datum.txt", &MSG_SIZE);
				#endif
			}

			//Stop timers
			hipEventRecord(stop,0);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&elapsed_gpu, start, stop);
			hipEventDestroy(start);
			hipEventDestroy(stop);

		#ifdef ITERATE_BLOCKS
			//Calculate results
			num_hashes = GDIMX*i;
			//block size, hashrate, hashes, execution time
			printf("%d, %.2f, %.0f, %.2f\n", i, num_hashes/(elapsed_gpu*1e-3), num_hashes, elapsed_gpu);
		}
		#endif
		//Copy nonce result back to host
		CUDA_SAFE_CALL(hipMemcpy(host_nonce, &device_nonce[0], 40 * sizeof(unsigned int), hipMemcpyDeviceToHost));

		/*	
			Post Processing
			Check the results of mining and print out debug information
		*/

		//Cuda Printf output
		hipDeviceSynchronize();
		cudaPrintfDisplay(stdout, false);
		cudaPrintfEnd();

		//Free memory on device
		CUDA_SAFE_CALL(hipFree(device_nonce));
		//CUDA_SAFE_CALL(hipFree(d_debug));
		
		//Output the results
		int count = 0;
		for (int i = 0; i < 40; i+=4) {
			if (host_nonce[i+3] %2 == 1) {
				count++;
			} else {
				break;
			}
		}
		printf("%i nonces found\n", count);
		if(host_nonce[3] % 2 == 1) {
			host_nonce[3]--;
			printf("Nonce found! %.8x ", host_nonce[0]);
			printf("%.8x ", host_nonce[1]);
			printf("%.8x ", host_nonce[2]);
			printf("%.8x ", host_nonce[3]);
			printf("\n");
			store_nonce("./submit.txt", &host_nonce[0]);
			printf("Difficulty: " );
			for(int i=0; i<16; i++) {
				printf("%.2x", difficulty[i]);
			}
			printf("\n");
			compute_and_print_hash(data, host_nonce, MSG_SIZE);

			// data = get_file_data("./datum.txt", &MSG_SIZE);
			host_nonce[3] = 0;
		}
		// else {
		// 	printf("Nonce not found :(\n");
		// }

		#ifdef VERIFY_HASH
		break;
		#endif
	}
}

//Declare SHA-256 constants
__constant__ uint32_t k[64] = {
	0x428a2f98,0x71374491,0xb5c0fbcf,0xe9b5dba5,0x3956c25b,0x59f111f1,0x923f82a4,0xab1c5ed5,
	0xd807aa98,0x12835b01,0x243185be,0x550c7dc3,0x72be5d74,0x80deb1fe,0x9bdc06a7,0xc19bf174,
	0xe49b69c1,0xefbe4786,0x0fc19dc6,0x240ca1cc,0x2de92c6f,0x4a7484aa,0x5cb0a9dc,0x76f988da,
	0x983e5152,0xa831c66d,0xb00327c8,0xbf597fc7,0xc6e00bf3,0xd5a79147,0x06ca6351,0x14292967,
	0x27b70a85,0x2e1b2138,0x4d2c6dfc,0x53380d13,0x650a7354,0x766a0abb,0x81c2c92e,0x92722c85,
	0xa2bfe8a1,0xa81a664b,0xc24b8b70,0xc76c51a3,0xd192e819,0xd6990624,0xf40e3585,0x106aa070,
	0x19a4c116,0x1e376c08,0x2748774c,0x34b0bcb5,0x391c0cb3,0x4ed8aa4a,0x5b9cca4f,0x682e6ff3,
	0x748f82ee,0x78a5636f,0x84c87814,0x8cc70208,0x90befffa,0xa4506ceb,0xbef9a3f7,0xc67178f2
};

#define NONCE_VAL (gridDim.x*blockDim.x*blockIdx.y + blockDim.x*blockIdx.x + blockDim.x*gridDim.x*threadIdx.y + threadIdx.x)

__device__ void sha256_second_update_cuda(SHA256_RX *ctx)
{
	ctx->datalen = 32;
	ctx->bitlen = 0;
}

#define CUDA_EP0(x) (((x / 4) | (x * 1073741824)) ^ ((x / 8192) | (x * 524288)) ^ ((x / 4194304) | (x * 1024)))
#define CUDA_EP1(x) (((x / 64) | (x * 67108864)) ^ ((x / 2048) | (x * 2097152)) ^ ((x / 33554432) | (x * 128)))

#define TRANSFORM_BODY		\
		t1 = CH(e,f,g);		\
		t1 += h;			\
		h = g;				\
		t1 += CUDA_EP1(e);	\
		g = f;				\
		t1 += k[j];			\
		t2 = MAJ(a,b,c);	\
		f = e;				\
		t1 += m[i];			\
		e = d + t1;			\
		d = c;				\
		t2 += CUDA_EP0(a);	\
		c = b;				\
		b = a;				\
		a = t1 + t2;		\

__device__ void cuda_sha256_transform(SHA256_RX *ctx) {
	WORD a, b, c, d, e, f, g, h, t1, t2, m[16];
	int i, j;

	a = ctx->state[0];
	b = ctx->state[1];
	c = ctx->state[2];
	d = ctx->state[3];
	e = ctx->state[4];
	f = ctx->state[5];
	g = ctx->state[6];
	h = ctx->state[7];
	j = 0;
	for (i = 0; i < 16; ++i) {
		m[i] = ctx->data.word[i];

		TRANSFORM_BODY;
		
		j++;
	}
	for (i = 0; i < 7; ++i) {
		m[i] = SIG1(m[(i + 14) & 0xf]);
		m[i] += m[i + 9];
		m[i] += SIG0(m[i + 1]);
		m[i] += ctx->data.word[i];

		TRANSFORM_BODY;

		j++;
	}
	for (i; i < 15; ++i) {
		m[i] = SIG1(m[(i - 2)]);
		m[i] += m[((i + 9) & 0xf)];
		m[i] += SIG0(m[(i + 1)]);
		m[i] += ctx->data.word[i];

		TRANSFORM_BODY;

		j++;
	}
	for (i; i < 16; ++i) {
		m[i] = SIG1(m[(i - 2)]);
		m[i] += m[((i + 9) & 0xf)];
		m[i] += SIG0(m[(i + 1) & 0xf]);
		m[i] += ctx->data.word[i];

		TRANSFORM_BODY;

		j++;
	}
	for (i = 0; i < 7; ++i) {
		t1 = m[i];
		m[i] = SIG1(m[(i + 14) & 0xf]);
		m[i] += m[((i + 9))];
		m[i] += SIG0(m[(i + 1)]);
		m[i] += t1;

		TRANSFORM_BODY;

		j++;
	}
	for (i; i < 15; ++i) {
		t1 = m[i];
		m[i] = SIG1(m[(i - 2)]);
		m[i] += m[((i + 9) & 0xf)];
		m[i] += SIG0(m[(i + 1)]);
		m[i] += t1;

		TRANSFORM_BODY;

		j++;
	}
	for (i; i < 16; ++i) {
		t1 = m[i];
		m[i] = SIG1(m[(i - 2)]);
		m[i] += m[((i + 9) & 0xf)];
		m[i] += SIG0(m[(i + 1) & 0xf]);
		m[i] += t1;

		TRANSFORM_BODY;

		j++;
	}
	for (i = 0; i < 7; ++i) {
		t1 = m[i];
		m[i] = SIG1(m[(i + 14) & 0xf]);
		m[i] += m[((i + 9))];
		m[i] += SIG0(m[(i + 1)]);
		m[i] += t1;

		TRANSFORM_BODY;

		j++;
	}
	for (i; i < 15; ++i) {
		t1 = m[i];
		m[i] = SIG1(m[(i - 2)]);
		m[i] += m[((i + 9) & 0xf)];
		m[i] += SIG0(m[(i + 1)]);
		m[i] += t1;

		TRANSFORM_BODY;

		j++;
	}
	for (i; i < 16; ++i) {
		t1 = m[i];
		m[i] = SIG1(m[(i - 2)]);
		m[i] += m[((i + 9) & 0xf)];
		m[i] += SIG0(m[(i + 1) & 0xf]);
		m[i] += t1;

		TRANSFORM_BODY;

		j++;
	}
	ctx->state[0] += a;
	ctx->state[1] += b;
	ctx->state[2] += c;
	ctx->state[3] += d;
	ctx->state[4] += e;
	ctx->state[5] += f;
	ctx->state[6] += g;
	ctx->state[7] += h;
}

__device__ void cuda_sha256_init(SHA256_RX *ctx) {
	ctx->datalen = 0;
	ctx->bitlen = 0;
	ctx->state[0] = 0x6a09e667;
	ctx->state[1] = 0xbb67ae85;
	ctx->state[2] = 0x3c6ef372;
	ctx->state[3] = 0xa54ff53a;
	ctx->state[4] = 0x510e527f;
	ctx->state[5] = 0x9b05688c;
	ctx->state[6] = 0x1f83d9ab;
	ctx->state[7] = 0x5be0cd19;
}

__device__ void cuda_sha256_first_update(SHA256_RX *ctx) {
	for (int i = 0; i < 27; ++i) {
		ctx->data.word[i] = ENDIAN_SWAP_32(ctx->data.word[i]);
	}							
	cuda_sha256_transform(ctx);
	ctx->bitlen = 512;
	ctx->datalen = ctx->datalen - 64;
	for (int i = 16; i < 27; ++i) {
		ctx->data.word[i-16] = ctx->data.word[i];
	}
}

__device__ void cuda_sha256_first_pad(SHA256_RX *ctx) {
	WORD i;

	i = 4 * (ctx->datalen / 4);

	ctx->data.byte[i + 3 - (ctx->datalen % 4)] = 0x80;
	for (int j = 2; i + j >= ctx->datalen; j--) {
		ctx->data.byte[i + j - (ctx->datalen % 4)] = 0;
	}
	// ctx->data.byte[i + 2 - (ctx->datalen % 4)] = 0;
	// ctx->data.byte[i + 1 - (ctx->datalen % 4)] = 0;
	// ctx->data.byte[i - (ctx->datalen % 4)] = 0;

	i += 4;

	while (i < 60)
		ctx->data.byte[i++] = 0x00;

	// Store value of l
	ctx->bitlen += ctx->datalen * 8;
	ctx->data.byte[60] = ctx->bitlen;
	ctx->data.byte[61] = ctx->bitlen >> 8;
	ctx->data.byte[62] = 0;
	ctx->data.byte[63] = 0;
}

__device__ void cuda_sha256_second_pad(SHA256_RX *ctx) {
	WORD i;

	i = ctx->datalen;

	ctx->data.byte[i++] = 0x00;
	ctx->data.byte[i++] = 0x00;
	ctx->data.byte[i++] = 0x00;
	ctx->data.byte[i++] = 0x80;
	while (i < 60)
		ctx->data.byte[i++] = 0x00;
	ctx->bitlen += ctx->datalen * 8;
	ctx->data.byte[60] = ctx->bitlen;
	ctx->data.byte[61] = ctx->bitlen >> 8;
	ctx->data.byte[62] = 0;
	ctx->data.byte[63] = 0;
}

__device__ void cuda_sha256_first_final(SHA256_RX *ctx) {
	cuda_sha256_first_pad(ctx);
	cuda_sha256_transform(ctx);
}

__device__ void cuda_sha256_second_final(SHA256_RX *ctx) {
	cuda_sha256_second_pad(ctx);
	cuda_sha256_transform(ctx);
}

__global__ void kernel_sha256d(unsigned int *nonce, void *debug) {
	int i, j;
	
    SHA256_RX ctx;

	// Synchronized load data to shared memory
	__shared__ uint32_t shared_k[64];
	__shared__ DATA shared_data;
	__shared__ DIFFICULTY shared_difficulty;
	__shared__ WORD msglen;
	__shared__ unsigned long NLOOPS;
	// __shared__ WORD shared_nonce [GDIMX*GDIMY*3];
	#ifndef VERIFY_HASH
	i = threadIdx.y * GDIMX + threadIdx.x;
	if (i < 64) {
		if (i < 16) {
			if (i == 0) {
				msglen = device_msg_len;
				NLOOPS = nloops;
			}
			shared_difficulty.byte[i] = device_difficulty[i];
		}
		shared_k[i] = k[i];
		shared_data.byte[i] = device_data[i];
	} else if (i < 105)
	{
		shared_data.byte[i] = device_data[i];
	}
	__syncthreads();

	// Set the local nonce
	ctx.nonce[0] = nonce[0];
	ctx.nonce[1] = NONCE_VAL;
	ctx.nonce[2] = nonce[2];
	ctx.nonce[3] = nonce[3];
	#else
	for (int t = 0; t < 64; t++) {
		if (t < 16) {
			shared_difficulty.byte[t] = device_difficulty[t];
		}
		shared_k[t] = k[t];
		shared_data.byte[t] = device_data[t];
	} 
	for (int t = 64; t < 105; t++)
	{
		shared_data.byte[t] = device_data[t];
	}
	
	// Set the local nonce
	ctx.nonce[0] = nonce[0];
	ctx.nonce[1] = 0;
	ctx.nonce[2] = 0;
	ctx.nonce[3] = 16777216;
	
	NLOOPS = nloops;
	#endif

	// Copy data to local registers
	for (i = 0; i < 64; ++i) {
		ctx.k[i] = shared_k[i];
	}

	// Initialize bitlen to 0
	ctx.bitlen = 0;

	for (int loop = 0; loop < NLOOPS; loop ++) {

		#ifndef VERIFY_HASH
		ctx.nonce[3] = 2 * loop;
		#endif

		ctx.data.word[0] = shared_data.word[0];
		ctx.data.word[1] = ctx.nonce[0];
		ctx.data.word[2] = ctx.nonce[1];
		ctx.data.word[3] = ctx.nonce[2];
		ctx.data.word[4] = ctx.nonce[3];
		for (i = 5 ; i < 27; ++i) {
			ctx.data.word[i] = shared_data.word[i];
		}
			#ifdef VERIFY_HASH
			// get the message length
			msglen = device_msg_len;
			unsigned int *ref_hash = (unsigned int *) debug;
			cuPrintf("--Cuda--\n");
			cuPrintf("Initial Data: ");
			for(i=0; i<msglen; i++) {
				cuPrintf("%.2x", shared_data.byte[i]);
			}
			cuPrintf("\n");
			cuPrintf("CTX Data: ");
			for(i=0; i<msglen; i++) {
				cuPrintf("%.2x", ctx.data.byte[i]);
			}
			cuPrintf("\n");
			cuPrintf("NONCE: ");
			for(i=0; i<msglen; i++) {
				cuPrintf("%.2x", ctx.data.byte[i]);
			}
			cuPrintf("\n");
			#endif
			
		// // For debugging
		// for (i = 0 ; i < 26; ++i) {
		// 	shared_data.word[i] = ctx.data.word[i];
		// }

		cuda_sha256_init(&ctx);
		ctx.datalen = msglen;
			#ifdef VERIFY_HASH
			cuPrintf("ctx.datalen: %i\n", 4*((ctx.datalen - 64)/4));
			cuPrintf("1. init state: ");
			for(int i=0; i<8; i++) {
				cuPrintf("%.8x ", ENDIAN_SWAP_32(ctx.state[i]));
			}
			cuPrintf("\n");
			#endif
			
		// First value update
		cuda_sha256_first_update(&ctx);
			#ifdef VERIFY_HASH
			cuPrintf("2. update state: ");
			for(int i=0; i<8; i++) {
				cuPrintf("%.8x ", ENDIAN_SWAP_32(ctx.state[i]));
			}
			cuPrintf("\n");
			cuPrintf("Pad: ");
			for(int i=0; i<16; i++) {
				for (int j=(i+1)*4 - 1; j>=i*4; j--) {
					cuPrintf("%.2x ", ctx.data.byte[j]);
					// cuPrintf("%i ", j);
				}
			}
			cuPrintf("\n");
			#endif
		cuda_sha256_first_pad(&ctx);
			#ifdef VERIFY_HASH
			cuPrintf("Pad: ");
			for(int i=0; i<16; i++) {
				for (int j=(i+1)*4 - 1; j>=i*4; j--) {
					cuPrintf("%.2x ", ctx.data.byte[j]);
					// cuPrintf("%i ", j);
				}
			}
			cuPrintf("\n");
			#endif
		cuda_sha256_transform(&ctx);
		// cuda_sha256_first_final(&ctx);
		for (int i = 0; i < 8; ++i) {
			ctx.data.word[i] = ctx.state[i];
		}																	
			// FINAL;
			// sha256_final_cuda(&ctx);
			#ifdef VERIFY_HASH
			cuPrintf("3. final state: ");
			for(int i=0; i<8; i++) {
				cuPrintf("%.8x ", ENDIAN_SWAP_32(ctx.state[i]));
			}
			cuPrintf("\n");
			#endif
		cuda_sha256_init(&ctx);
		ctx.datalen = 32;
			#ifdef VERIFY_HASH
			cuPrintf("4. init state: ");
			for(int i=0; i<8; i++) {
				cuPrintf("%.8x ", ENDIAN_SWAP_32(ctx.state[i]));
			}
			cuPrintf("\n");
			#endif
		sha256_second_update_cuda(&ctx);
			#ifdef VERIFY_HASH
			cuPrintf("5. update state: ");
			for(int i=0; i<8; i++) {
				cuPrintf("%.8x ", ENDIAN_SWAP_32(ctx.state[i]));
			}
			cuPrintf("\n");
			cuPrintf("Pad: ");
			for(int i=0; i<64; i++) {
				cuPrintf("%.2x ", ctx.data.byte[i]);
			}
			cuPrintf("\n");
			#endif
		cuda_sha256_second_final(&ctx);
			#ifdef VERIFY_HASH
			cuPrintf("6. final state: ");
			for(int i=0; i<8; i++) {
				cuPrintf("%.8x ", ENDIAN_SWAP_32(ctx.state[i]));
			}
			cuPrintf("\n");
			cuPrintf("Final Hash: ");
			for(i=0; i<8; i++) {
				cuPrintf("%.8x ", ctx.state[i]);
			}
			cuPrintf("\n");
			#endif

			#ifdef VERIFY_HASH
			cuPrintf("Difficulty\n");
			for(i=0; i<16; i++) {
				cuPrintf("%.2x", shared_difficulty.byte[i]);
			}
			#endif

		for (i = 0; i < 8; ++i) {
			ctx.data.word[i] = ENDIAN_SWAP_32(ctx.state[i]);
		}

		i=0;
		while(ctx.data.byte[i] == shared_difficulty.byte[i])
			i++;
		
		
		if(ctx.data.byte[i] < shared_difficulty.byte[i]) {
			//Synchronization Issue
			//Kind of a hack but it really doesn't matter which nonce
			//is written to the output, they're all winners :)
			//Further it's unlikely to even find a nonce let alone 2
			for (i = 0; i < 40; i+=4) {
				if (nonce[i+3] == 0) {
					nonce[i] = ctx.nonce[0];
					nonce[i+1] = ctx.nonce[1];
					nonce[i+2] = ctx.nonce[2];
					nonce[i+3] = ctx.nonce[3] + 1;
					break;
				}
			}
		}
	}
}

#ifndef VERIFY_HASH
std::vector<std::string> mine_cuda(py::bytes datum, unsigned int zeros, unsigned long NLOOPS) {
	const std::string data(datum);
	unsigned long MSG_SIZE = data.length();

	dim3 DimGrid(GDIMX,GDIMY);
	dim3 DimBlock(BDIMX,1);

	// For debugger
	unsigned char debug[32];
	unsigned char *d_debug;

	//Setup host nonce
	unsigned int * host_nonce = new unsigned int[40];
	memset(host_nonce, 0, sizeof(unsigned int) * 40);
	unsigned int * device_nonce = new unsigned int[40];
	memset(device_nonce, 0, sizeof(unsigned int) * 40);

	// Initialize host nonce
	std::mt19937 mt{ std::random_device{}() };
	host_nonce[0] = *((unsigned int *) (data.data() + 4)); // unique pool part nonce
	host_nonce[1] = 0; 									   // grid location nonce
	host_nonce[2] = mt();								   // random nonce
	host_nonce[3] = 0;									   // increment nonce

	// Send nonce to device
	CUDA_SAFE_CALL(hipMalloc((void **) &device_nonce, 40 * sizeof(unsigned int)));
	CUDA_SAFE_CALL(hipMemcpy(device_nonce, &host_nonce[0], 40 * sizeof(unsigned int), hipMemcpyHostToDevice));

	//Decodes and stores the difficulty in a 16-byte array for convenience
	unsigned char * difficulty = set_tuna_difficulty(65535, zeros);

	//Send data to device
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(device_data), &data[0], 105));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(device_difficulty), &difficulty[0], 16));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(device_msg_len), &MSG_SIZE, 4));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(nloops), &NLOOPS, 4));

	//Launch Kernel
	kernel_sha256d<<<DimGrid, DimBlock>>>(device_nonce, (void *) d_debug);

	//Copy nonce result back to host
	CUDA_SAFE_CALL(hipMemcpy(host_nonce, &device_nonce[0], 40 * sizeof(unsigned int), hipMemcpyDeviceToHost));
	
	//Free memory on device
	CUDA_SAFE_CALL(hipFree(device_nonce));
	//CUDA_SAFE_CALL(hipFree(d_debug));

	std::vector<std::string> output;

	for (int i = 0; i < 40; i+=4) {
		if (host_nonce[i+3] %2 == 1) {
			std::stringstream stream;
			host_nonce[i+3]--;
			for (int j = i; j < i+4; ++j) {
				stream << std::setfill('0') << std::setw(8) << std::hex << ENDIAN_SWAP_32(host_nonce[j]);
			}
			output.push_back(stream.str());
		} else {
			break;
		}
	}

	return output;
}

PYBIND11_MODULE(gpu_library, m) {
    m.doc() = "Fortuna miner...for cuda."; // optional module docstring

    m.def("mine_cuda", &mine_cuda, R"pbdoc(
        Mine using cuda.
    )pbdoc");
}
#endif